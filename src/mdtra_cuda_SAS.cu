#include "hip/hip_runtime.h"
/***************************************************************************
* Copyright (C) 2011-2017 Alexander V. Popov.
* 
* This file is part of Molecular Dynamics Trajectory 
* Reader & Analyzer (MDTRA) source code.
* 
* MDTRA source code is free software; you can redistribute it and/or 
* modify it under the terms of the GNU General Public License as 
* published by the Free Software Foundation; either version 2 of 
* the License, or (at your option) any later version.
* 
* MDTRA source code is distributed in the hope that it will be 
* useful, but WITHOUT ANY WARRANTY; without even the implied 
* warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
* See the GNU General Public License for more details.
* 
* You should have received a copy of the GNU General Public License
* along with this program; if not, write to the Free Software 
* Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA 02110-1301 USA
***************************************************************************/
#include "mdtra_main.h"
#include "mdtra_math.h"
#include "mdtra_cuda.h"
#include "mdtra_pdb_flags.h"
#include "mdtra_pdb.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_math_constants.h>

// within ideal capability, run 512 threads per block, 256 otherwise
#define CUDA_IDEAL_COMPUTE_CAPABILITY	1.2f

// this value depends on max. accuracy
// 642 dots for accuracy level = 3
#define CUDA_SAS_MAX_DOTS				642
#define CUDA_SAS_MAX_DOTS_ALIGN			768

#define CUDA_SAS_ATOM_SIZE				(sizeof(float)*4 + sizeof(int))

__constant__ float cudaSASDots[CUDA_SAS_MAX_DOTS*3];
__constant__ int cudaSASNumDots;

static void* cudaHostMemory = NULL;
static void* cudaDeviceAtoms = NULL;
static float* cudaDeviceResult = NULL;
static int cudaMaxAtoms = 0;
static int cudaMaxThreads = 0;
static int sasExcludeMask = 0;
static bool s_cudaError = false;

__device__ float calc_SAS_surface( float radius, int buried )
{
	float areaTotal = 4.0f * HIP_PI_F * radius * radius;
	float areaRatio = areaTotal * ((float)buried / cudaSASNumDots);
	return areaTotal - areaRatio;
}

__device__ float calc_OCC_surface( float radius, int buried )
{
	float areaTotal = 4.0f * HIP_PI_F * radius * radius;
	return areaTotal * ((float)buried / cudaSASNumDots);
}

__device__ int process_SAS_atom_pair( float x0, float y0, float z0, float r0, float x1, float y1, float z1, float r1, int* dotinfo )
{
	float diff[3];
	float delta;
	float origin[3];

	// check for sphere intersection
	diff[0] = x1 - x0;
	diff[1] = y1 - y0;
	diff[2] = z1 - z0;
	delta = diff[0]*diff[0] + diff[1]*diff[1] + diff[2]*diff[2];

	if ( delta >= (r0+r1)*(r0+r1) )
		return 1;

	int accessible = 0;
	float r1sq = r1 * r1;

	// check each dot against this neighbour
	const float* pDotCoords = cudaSASDots;
	for ( int k = 0; k < cudaSASNumDots; k++, pDotCoords += 3 ) {
		// this dot has already been discarded
		if ( dotinfo[k] != 0 )
			continue;

		// get dot origin relative to the neighbour
		origin[0] = pDotCoords[0] * r0 + diff[0];
		origin[1] = pDotCoords[1] * r0 + diff[1];
		origin[2] = pDotCoords[2] * r0 + diff[2];
		delta = origin[0]*origin[0] + origin[1]*origin[1] + origin[2]*origin[2];

		// check if it is inside the sphere
		if ( delta < r1sq ) {
			//buried
			dotinfo[k] = 1;
			continue;
		}

		// accessible
		accessible = 1;
	}

	return accessible;
}

__device__ int process_OCC_atom_pair( float x0, float y0, float z0, float r0, float x1, float y1, float z1, float r1, int occluder, int* dotinfo1, int* dotinfo2 )
{
	float diff[3];
	float delta;
	float origin[3];

	// check for sphere intersection
	diff[0] = x1 - x0;
	diff[1] = y1 - y0;
	diff[2] = z1 - z0;
	delta = diff[0]*diff[0] + diff[1]*diff[1] + diff[2]*diff[2];

	if ( delta >= (r0+r1)*(r0+r1) )
		return 1;

	int accessible = 0;
	float r1sq = r1 * r1;

	// check each dot against this neighbour
	const float* pDotCoords = cudaSASDots;
	for ( int k = 0; k < cudaSASNumDots; k++, pDotCoords += 3 ) {
		// this dot has already been discarded
		if ( dotinfo2[k] != 0 )
			continue;
		if ( occluder && ( dotinfo1[k] != 0 ) )
			continue;

		// get dot origin relative to the neighbour
		origin[0] = pDotCoords[0] * r0 + diff[0];
		origin[1] = pDotCoords[1] * r0 + diff[1];
		origin[2] = pDotCoords[2] * r0 + diff[2];
		delta = origin[0]*origin[0] + origin[1]*origin[1] + origin[2]*origin[2];

		// check if it is inside the sphere
		if ( delta < r1sq ) {
			//buried
			dotinfo1[k] = 1;
			if ( !occluder ) {
				dotinfo2[k] = 1;
				continue;
			}
		}

		// accessible
		accessible = 1;
	}

	return accessible;
}

__global__ void kernel_SAS( void* pAtoms, int iStride, float* outData ) 
{
	__shared__ int dotinfo[CUDA_SAS_MAX_DOTS_ALIGN];

	int firstAtomIndex = blockIdx.x;
	int secondAtomIndex = threadIdx.x;
	int numAtoms = gridDim.x;

	float* pAtX = (float*)pAtoms;
	float* pAtY = pAtX + iStride;
	float* pAtZ = pAtY + iStride;
	float* pAtR = pAtZ + iStride;
	int*   pAtFlags = (int*)(pAtR + iStride);

	// check if we take this atom into account
	if ( pAtFlags[firstAtomIndex] == 0 ) {
		// nothing to calculate at this block
		if ( threadIdx.x == 0 ) outData[firstAtomIndex] = 0;
		return;
	}

	// clear shared memory
	int currentDot = threadIdx.x;
	while ( currentDot < cudaSASNumDots ) {
		dotinfo[currentDot] = 0;
		currentDot += blockDim.x;
	}
	__syncthreads();

	// get first atom's xyzr
	float x0 = pAtX[firstAtomIndex];
	float y0 = pAtY[firstAtomIndex];
	float z0 = pAtZ[firstAtomIndex];
	float r0 = pAtR[firstAtomIndex];

	// fill shared memory
	while ( secondAtomIndex < numAtoms ) {
		if ( firstAtomIndex != secondAtomIndex ) {
			float x1 = pAtX[secondAtomIndex];
			float y1 = pAtY[secondAtomIndex];
			float z1 = pAtZ[secondAtomIndex];
			float r1 = pAtR[secondAtomIndex];
			if ( !process_SAS_atom_pair( x0, y0, z0, r0, x1, y1, z1, r1, dotinfo ) )
				break;
		}
		secondAtomIndex += blockDim.x;
	}
	__syncthreads();

	// reduction
	int i = blockDim.x;
	while ( i != 0 ) {
		if ( cudaSASNumDots > i ) {
			if ( threadIdx.x < i && threadIdx.x < cudaSASNumDots-i )
				dotinfo[threadIdx.x] += dotinfo[threadIdx.x+i];
			__syncthreads();
		}
		i >>= 1;
	}

	// return surface for atom
	if ( threadIdx.x == 0 )
		outData[firstAtomIndex] = calc_SAS_surface( pAtR[firstAtomIndex], dotinfo[0] );
}

__global__ void kernel_OCC( void* pAtoms, int iStride, float* outData ) 
{
	__shared__ int dotinfo1[CUDA_SAS_MAX_DOTS_ALIGN];
	__shared__ int dotinfo2[CUDA_SAS_MAX_DOTS_ALIGN];

	int firstAtomIndex = blockIdx.x;
	int secondAtomIndex = threadIdx.x;
	int numAtoms = gridDim.x;

	float* pAtX = (float*)pAtoms;
	float* pAtY = pAtX + iStride;
	float* pAtZ = pAtY + iStride;
	float* pAtR = pAtZ + iStride;
	int*   pAtFlags = (int*)(pAtR + iStride);

	// check if we take this atom into account
	if ( !(pAtFlags[firstAtomIndex] & PDB_FLAG_SAS) ||
		  (pAtFlags[firstAtomIndex] & PDB_FLAG_OCCLUDER) ) {
		if ( threadIdx.x == 0 ) outData[firstAtomIndex] = 0;
		return;
	}

	// clear shared memory
	int currentDot = threadIdx.x;
	while ( currentDot < cudaSASNumDots ) {
		dotinfo1[currentDot] = 0;
		dotinfo2[currentDot] = 0;
		currentDot += blockDim.x;
	}
	__syncthreads();

	// get first atom's xyzr
	float x0 = pAtX[firstAtomIndex];
	float y0 = pAtY[firstAtomIndex];
	float z0 = pAtZ[firstAtomIndex];
	float r0 = pAtR[firstAtomIndex];

	// fill shared memory
	while ( secondAtomIndex < numAtoms ) {
		if ( firstAtomIndex != secondAtomIndex ) {
			float x1 = pAtX[secondAtomIndex];
			float y1 = pAtY[secondAtomIndex];
			float z1 = pAtZ[secondAtomIndex];
			float r1 = pAtR[secondAtomIndex];
			int occluder = pAtFlags[secondAtomIndex] & PDB_FLAG_OCCLUDER;
			if ( !process_OCC_atom_pair( x0, y0, z0, r0, x1, y1, z1, r1, occluder, dotinfo1, dotinfo2 ) )
				break;
		}
		secondAtomIndex += blockDim.x;
	}
	__syncthreads();
	 
	// reduction
	int i = blockDim.x;
	while ( i != 0 ) {
		if ( cudaSASNumDots > i ) {
			if ( threadIdx.x < i && threadIdx.x < cudaSASNumDots-i ) {
				dotinfo1[threadIdx.x] += dotinfo1[threadIdx.x+i];
				dotinfo2[threadIdx.x] += dotinfo2[threadIdx.x+i];
			}
			__syncthreads();
		}
		i >>= 1;
	}

	// return surface for atom
	if ( threadIdx.x == 0 )
		outData[firstAtomIndex] = calc_OCC_surface( pAtR[firstAtomIndex], dotinfo1[0] - dotinfo2[0] );
}

__host__ void alloc_memory( int numAtoms )
{
	size_t atoms_size = numAtoms * CUDA_SAS_ATOM_SIZE * CountThreads();
	size_t result_size = numAtoms * sizeof(float) * CountThreads();

	MDTRA_CUDA_SAFE_CALL( hipHostMalloc( (void**)&cudaHostMemory, atoms_size ) );
	MDTRA_CUDA_SAFE_CALL( hipMalloc( (void**)&cudaDeviceAtoms, atoms_size ) );
	MDTRA_CUDA_SAFE_CALL( hipMalloc( (void**)&cudaDeviceResult, result_size ) );
}

__host__ void free_memory( void )
{
	if ( cudaHostMemory ) {
		MDTRA_CUDA_SAFE_CALL( hipHostFree( cudaHostMemory ) );
		cudaHostMemory = NULL;
	}
	if ( cudaDeviceAtoms ) {
		MDTRA_CUDA_SAFE_CALL( hipFree( cudaDeviceAtoms ) );
		cudaDeviceAtoms = NULL;
	}
	if ( cudaDeviceResult ) {
		MDTRA_CUDA_SAFE_CALL( hipFree( cudaDeviceResult ) );
		cudaDeviceResult = NULL;
	}

	cudaMaxAtoms = 0;
}

__host__ void ensure_memory_capacity( int numAtoms )
{
	if ( numAtoms > cudaMaxAtoms || CountThreads() > cudaMaxThreads) {
		ThreadLock();
		free_memory();
		if ( !cudaMaxAtoms ) cudaMaxAtoms = MDTRA_CUDA_ATOMS_INIT;
		while ( numAtoms > cudaMaxAtoms ) cudaMaxAtoms += MDTRA_CUDA_ATOMS_GROW;
		alloc_memory( cudaMaxAtoms );
		cudaMaxThreads = CountThreads();
		ThreadUnlock();
	}
}

__host__ void MDTRA_CUDA_InitSAS( const float* pDots, int numDots, dword excludeMask )
{
	s_cudaError = false;

	if ( cudaHostMemory )
		return;

	assert( numDots < (512*2) ); //for kernel reduction
	assert( numDots <= CUDA_SAS_MAX_DOTS );

	sasExcludeMask = excludeMask;

	MDTRA_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "cudaSASDots"), pDots, sizeof(float)*3*numDots ) );
	MDTRA_CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL( "cudaSASNumDots"), &numDots, sizeof(numDots) ) );

	ensure_memory_capacity( MDTRA_CUDA_ATOMS_INIT );
}

__host__ void MDTRA_CUDA_ShutdownSAS( void )
{
	free_memory();
}

__host__ float MDTRA_CUDA_CalculateSAS( int threadnum, const MDTRA_PDB_Atom *pAtoms, int numAtoms )
{
	if ( s_cudaError )
		return 0.0f;

	// check memory
	ensure_memory_capacity( numAtoms );

	// get device pointers
	void* cudaThreadDeviceAtoms = (float*)((char*)cudaDeviceAtoms + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* cudaThreadDeviceResult = (float*)((char*)cudaDeviceResult + numAtoms * sizeof(float) * threadnum);

	// build atoms on the host
	const MDTRA_PDB_Atom* pAt = pAtoms;
	float* cudaThreadHostMemory = (float*)((char*)cudaHostMemory + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* hostX = cudaThreadHostMemory;
	float* hostY = hostX + numAtoms;
	float* hostZ = hostY + numAtoms;
	float* hostR = hostZ + numAtoms;
	int* hostFlags = (int*)(hostR + numAtoms);
	int realNumAtoms = 0;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;
		*hostX = pAt->original_xyz[0];
		*hostY = pAt->original_xyz[1];
		*hostZ = pAt->original_xyz[2];
		*hostR = pAt->sasRadius;
		*hostFlags = (pAt->atomFlags & PDB_FLAG_SAS);
		realNumAtoms++;
		hostX++, hostY++, hostZ++, hostR++, hostFlags++;
	}

	// copy atoms to device
	assert( realNumAtoms <= MDTRA_CUDA_ATOMS_MAX );
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadDeviceAtoms, cudaThreadHostMemory, numAtoms*CUDA_SAS_ATOM_SIZE, hipMemcpyHostToDevice ) );

	// run kernel
	int kernelSize = ( g_flComputeVersionCUDA >= CUDA_IDEAL_COMPUTE_CAPABILITY ) ? 512 : 256;
	kernel_SAS<<<realNumAtoms,kernelSize>>>( cudaThreadDeviceAtoms, numAtoms, cudaThreadDeviceResult );

	// get result
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadHostMemory, cudaThreadDeviceResult, realNumAtoms*sizeof(float), hipMemcpyDeviceToHost ) );

	// final reduction
	for ( int i = 1; i < realNumAtoms; i++ )
		cudaThreadHostMemory[0] += cudaThreadHostMemory[i];

	return cudaThreadHostMemory[0];
}

__host__ void MDTRA_CUDA_CalculateSASPerResidue( int threadnum, const MDTRA_PDB_Atom *pAtoms, int numAtoms, float* pOutData )
{
	if ( s_cudaError )
		return;

	// check memory
	ensure_memory_capacity( numAtoms );

	// get device pointers
	void* cudaThreadDeviceAtoms = (float*)((char*)cudaDeviceAtoms + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* cudaThreadDeviceResult = (float*)((char*)cudaDeviceResult + numAtoms * sizeof(float) * threadnum);

	// build atoms on the host
	const MDTRA_PDB_Atom* pAt = pAtoms;
	float* cudaThreadHostMemory = (float*)((char*)cudaHostMemory + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* hostX = cudaThreadHostMemory;
	float* hostY = hostX + numAtoms;
	float* hostZ = hostY + numAtoms;
	float* hostR = hostZ + numAtoms;
	int* hostFlags = (int*)(hostR + numAtoms);
	int realNumAtoms = 0;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;
		*hostX = pAt->original_xyz[0];
		*hostY = pAt->original_xyz[1];
		*hostZ = pAt->original_xyz[2];
		*hostR = pAt->sasRadius;
		*hostFlags = (pAt->atomFlags & PDB_FLAG_SAS);
		realNumAtoms++;
		hostX++, hostY++, hostZ++, hostR++, hostFlags++;
	}

	// copy atoms to device
	assert( realNumAtoms <= MDTRA_CUDA_ATOMS_MAX );
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadDeviceAtoms, cudaThreadHostMemory, numAtoms*CUDA_SAS_ATOM_SIZE, hipMemcpyHostToDevice ) );

	// run kernel
	int kernelSize = ( g_flComputeVersionCUDA >= CUDA_IDEAL_COMPUTE_CAPABILITY ) ? 512 : 256;
	kernel_SAS<<<realNumAtoms,kernelSize>>>( cudaThreadDeviceAtoms, numAtoms, cudaThreadDeviceResult );

	// get result
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadHostMemory, cudaThreadDeviceResult, realNumAtoms*sizeof(float), hipMemcpyDeviceToHost ) );

	// sum values per atom to residue surface buffer
	pAt = pAtoms;
	int iRSN = -1;
	int iRealIndex = 0;
	float* pflSAS = NULL;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;

		// check if changing residue
		if ( pAt->residuenumber != iRSN ) {
			iRSN = pAt->residuenumber;
			pflSAS = pOutData + pAt->residuenumber - 1;
		}

		*pflSAS += cudaThreadHostMemory[iRealIndex++];
	}
}

__host__ float MDTRA_CUDA_CalculateOcclusion( int threadnum, const MDTRA_PDB_Atom *pAtoms, int numAtoms )
{
	if ( s_cudaError )
		return 0.0f;

	// check memory
	ensure_memory_capacity( numAtoms );

	// get device pointers
	void* cudaThreadDeviceAtoms = (float*)((char*)cudaDeviceAtoms + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* cudaThreadDeviceResult = (float*)((char*)cudaDeviceResult + numAtoms * sizeof(float) * threadnum);

	// build atoms on the host
	const MDTRA_PDB_Atom* pAt = pAtoms;
	float* cudaThreadHostMemory = (float*)((char*)cudaHostMemory + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* hostX = cudaThreadHostMemory;
	float* hostY = hostX + numAtoms;
	float* hostZ = hostY + numAtoms;
	float* hostR = hostZ + numAtoms;
	int* hostFlags = (int*)(hostR + numAtoms);
	int realNumAtoms = 0;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;
		*hostX = pAt->original_xyz[0];
		*hostY = pAt->original_xyz[1];
		*hostZ = pAt->original_xyz[2];
		*hostR = pAt->sasRadius;
		*hostFlags = pAt->atomFlags;
		realNumAtoms++;
		hostX++, hostY++, hostZ++, hostR++, hostFlags++;
	}

	// copy atoms to device
	assert( realNumAtoms <= MDTRA_CUDA_ATOMS_MAX );
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadDeviceAtoms, cudaThreadHostMemory, numAtoms*CUDA_SAS_ATOM_SIZE, hipMemcpyHostToDevice ) );

	// run kernel
	int kernelSize = ( g_flComputeVersionCUDA >= CUDA_IDEAL_COMPUTE_CAPABILITY ) ? 512 : 256;
	kernel_OCC<<<realNumAtoms,kernelSize>>>( cudaThreadDeviceAtoms, numAtoms, cudaThreadDeviceResult );

	// get result
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadHostMemory, cudaThreadDeviceResult, realNumAtoms*sizeof(float), hipMemcpyDeviceToHost ) );

	// final reduction
	for ( int i = 1; i < realNumAtoms; i++ )
		cudaThreadHostMemory[0] += cudaThreadHostMemory[i];

	return cudaThreadHostMemory[0];
}

__host__ void MDTRA_CUDA_CalculateOcclusionPerResidue( int threadnum, const MDTRA_PDB_Atom *pAtoms, int numAtoms, float* pOutData )
{
	if ( s_cudaError )
		return;

	// check memory
	ensure_memory_capacity( numAtoms );

	// get device pointers
	void* cudaThreadDeviceAtoms = (float*)((char*)cudaDeviceAtoms + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* cudaThreadDeviceResult = (float*)((char*)cudaDeviceResult + numAtoms * sizeof(float) * threadnum);

	// build atoms on the host
	const MDTRA_PDB_Atom* pAt = pAtoms;
	float* cudaThreadHostMemory = (float*)((char*)cudaHostMemory + numAtoms * CUDA_SAS_ATOM_SIZE * threadnum);
	float* hostX = cudaThreadHostMemory;
	float* hostY = hostX + numAtoms;
	float* hostZ = hostY + numAtoms;
	float* hostR = hostZ + numAtoms;
	int* hostFlags = (int*)(hostR + numAtoms);
	int realNumAtoms = 0;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;
		*hostX = pAt->original_xyz[0];
		*hostY = pAt->original_xyz[1];
		*hostZ = pAt->original_xyz[2];
		*hostR = pAt->sasRadius;
		*hostFlags = pAt->atomFlags;
		realNumAtoms++;
		hostX++, hostY++, hostZ++, hostR++, hostFlags++;
	}

	// copy atoms to device
	assert( realNumAtoms <= MDTRA_CUDA_ATOMS_MAX );
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadDeviceAtoms, cudaThreadHostMemory, numAtoms*CUDA_SAS_ATOM_SIZE, hipMemcpyHostToDevice ) );

	// run kernel
	int kernelSize = ( g_flComputeVersionCUDA >= CUDA_IDEAL_COMPUTE_CAPABILITY ) ? 512 : 256;
	kernel_OCC<<<realNumAtoms,kernelSize>>>( cudaThreadDeviceAtoms, numAtoms, cudaThreadDeviceResult );

	// get result
	MDTRA_CUDA_SAFE_CALL( hipMemcpy( cudaThreadHostMemory, cudaThreadDeviceResult, realNumAtoms*sizeof(float), hipMemcpyDeviceToHost ) );

	// sum values per atom to residue surface buffer
	pAt = pAtoms;
	int iRSN = -1;
	int iRealIndex = 0;
	float* pflSAS = NULL;

	for ( int i = 0; i < numAtoms; i++, pAt++ ) {
		if ( pAt->atomFlags & sasExcludeMask )
			continue;

		// check if changing residue
		if ( pAt->residuenumber != iRSN ) {
			iRSN = pAt->residuenumber;
			pflSAS = pOutData + pAt->residuenumber - 1;
		}

		*pflSAS += cudaThreadHostMemory[iRealIndex++];
	}
}